#include <cmath>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 64
#define RANGE 10

__global__ void kernel(float * a, float * b, int n){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    float result;
    result = 0.8*(b[(i-1)+n*j+n*n*k]+b[(i+1)+n*j+n*n*k]+b[i+n*(j-1)+n*n*k]+b[i+n*(j+1)+n*n*k]+b[i+n*j+n*n*(k-1)]+b[i+n*j+n*n*(k+1)]);
    a[i+n*j+n*n*k]=(float)result;
    //atomicAdd(&a[i],1);
    printf("%f\n",result);
    //a[i][j][k]=0.8*(b[i-1][j][k]+b[i+1][j][k]+b[i][j-1][k]+b[i][j+1][k]+b[i][j][k-1]+b[i][j][k+1]);
}

float* q2(float * b){
    float* d_a;
    float* d_b;
    //int* d_n;
    hipError_t cudaStatus;
    // set up a on device
    cudaStatus = hipMalloc(&d_a, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not allocated.\n");
    cudaStatus = hipMemset(d_a, 0, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not memset.\n");
    // set up b on device
    cudaStatus = hipMalloc(&d_b, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_b not allocated.\n");
    cudaStatus = hipMemcpy(d_b, b, (N*N*N)*sizeof(float), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_b not copied.\n");
    // set up n on device
    /*cudaStatus = cudaMalloc((void**)&d_n, sizeof(int));
    if(cudaStatus != cudaSuccess)
        printf("d_n not allocated.\n");
    int n = N;
    cudaStatus = cudaMemcpy(d_n, &n, sizeof(int), cudaMemcpyHostToDevice);
    if(cudaStatus != cudaSuccess)
        printf("d_n not copied.\n");*/
    // finally, run on gpu
    int d = (int)sqrt(N);
    dim3 grid(d,d,d);
    dim3 block(d,d,d);
    kernel <<< grid, block >>> (d_a, d_b, N);
    hipDeviceSynchronize();
    // copy result back to host
    float * a = (float *)malloc(N*N*N*sizeof(float));
    cudaStatus = hipMemcpy(a, d_a, (N*N*N)*sizeof(float), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess)
        printf("a not returned to host.\n");
    hipFree(d_a); hipFree(d_b);
    return a;
}

int main(int argc, char* argv[]){
    float * b;
    b = (float *)malloc(N*N*N*sizeof(float));
    for(int i=0; i<N; i++) 
        for(int j=0; j<N; j++)
            for(int k=0; k<N; k++)
                b[i+j+k]= static_cast <float> (rand())/(static_cast <float> (RAND_MAX/RANGE));
    float * a;
    a = q2(b);
    for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            for(int k=0; k<N; k++)
                if(a[i+j+k]>0.) printf("[%d,%d,%d] : %f\n", i+1, j+1, k+1, a[i+j+k]); 
    return 0;
}
