#include <cmath>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 64
#define RANGE 10
#define PRINT_RESULT 0 /* Set 1 to print resulting array */
#define PRINT_RUNTIME 1 /* Set 1 to print runtime */

__global__ void tiled(float * a, float * b, int n){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    float result;
    result = 0.8*(b[(i-1)+n*j+n*n*k]+b[(i+1)+n*j+n*n*k]+b[i+n*(j-1)+n*n*k]+b[i+n*(j+1)+n*n*k]+b[i+n*j+n*n*(k-1)]+b[i+n*j+n*n*(k+1)]);
    a[i+n*j+n*n*k]=(float)result;
}

__global__ void tiled_shared_memory(float * a, float * b, int n){
    __shared__ float tmp[8*8*8];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    int block_location = threadIdx.x+threadIdx.y+threadIdx.z;
    float result;
    result = 0.8*(b[(i-1)+n*j+n*n*k]+b[(i+1)+n*j+n*n*k]+b[i+n*(j-1)+n*n*k]+b[i+n*(j+1)+n*n*k]+b[i+n*j+n*n*(k-1)]+b[i+n*j+n*n*(k+1)]);
    tmp[block_location] = (float)result;
    __syncthreads();
    a[i+n*j+n*n*k] = tmp[block_location];
}

__global__ void nontiled(float * a, float * b, int n){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float result;
    for(int j=1; j<n-1; j++){
        for(int k=1; k<n-1; k++){
            result = 0.8*(b[(i-1)+n*j+n*n*k]+b[(i+1)+n*j+n*n*k]+b[i+n*(j-1)+n*n*k]+b[i+n*(j+1)+n*n*k]+b[i+n*j+n*n*(k-1)]+b[i+n*j+n*n*(k+1)]);
            a[i+n*j+n*n*k]=(float)result;
        }
    }
}

float* q2_tiled(float * b){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* d_a;
    float* d_b;
    int* d_n;
    hipError_t cudaStatus;
    // set up a on device
    cudaStatus = hipMalloc(&d_a, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not allocated.\n");
    cudaStatus = hipMemset(d_a, 0, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not memset.\n");
    // set up b on device
    cudaStatus = hipMalloc(&d_b, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_b not allocated.\n");
    cudaStatus = hipMemcpy(d_b, b, (N*N*N)*sizeof(float), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_b not copied.\n");
    // set up n on device
    cudaStatus = hipMalloc((void**)&d_n, sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("d_n not allocated.\n");
    int n = N;
    cudaStatus = hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_n not copied.\n");
    // finally, run on gpu
    int d = (int)sqrt(N);
    dim3 grid(d,d,d);
    dim3 block(d,d,d);
    hipEventRecord(start);
    tiled <<< grid, block >>> (d_a, d_b, N);
    hipEventRecord(stop);
    // copy result back to host
    float * a = new float[N*N*N+1];
    cudaStatus = hipMemcpy(a, d_a, (N*N*N)*sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(PRINT_RUNTIME==1) printf("Tiled with shared memory took %f milliseconds.\n",milliseconds);
    if(cudaStatus != hipSuccess)
        printf("a not returned to host.\n");
    hipFree(d_a); hipFree(d_b);
    return a;
}

float* q2_tiled_shared_memory(float * b){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* d_a;
    float* d_b;
    int* d_n;
    hipError_t cudaStatus;
    // set up a on device
    cudaStatus = hipMalloc(&d_a, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not allocated.\n");
    cudaStatus = hipMemset(d_a, 0, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not memset.\n");
    // set up b on device
    cudaStatus = hipMalloc(&d_b, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_b not allocated.\n");
    cudaStatus = hipMemcpy(d_b, b, (N*N*N)*sizeof(float), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_b not copied.\n");
    // set up n on device
    cudaStatus = hipMalloc((void**)&d_n, sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("d_n not allocated.\n");
    int n = N;
    cudaStatus = hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_n not copied.\n");
    // finally, run on gpu
    int d = (int)sqrt(N);
    dim3 grid(d,d,d);
    dim3 block(d,d,d);
    hipEventRecord(start);
    tiled_shared_memory <<< grid, block >>> (d_a, d_b, N);
    hipEventRecord(stop);
    // copy result back to host
    float * a = new float[N*N*N+1];
    cudaStatus = hipMemcpy(a, d_a, (N*N*N)*sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(PRINT_RUNTIME==1) printf("Tiled took %f milliseconds.\n",milliseconds);
    if(cudaStatus != hipSuccess)
        printf("a not returned to host.\n");
    hipFree(d_a); hipFree(d_b);
    return a;
}

float* q2_nontiled(float * b){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* d_a;
    float* d_b;
    int* d_n;
    hipError_t cudaStatus;
    // set up a on device
    cudaStatus = hipMalloc(&d_a, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not allocated.\n");
    cudaStatus = hipMemset(d_a, 0, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_a not memset.\n");
    // set up b on device
    cudaStatus = hipMalloc(&d_b, (N*N*N)*sizeof(float));
    if(cudaStatus != hipSuccess)
        printf("d_b not allocated.\n");
    cudaStatus = hipMemcpy(d_b, b, (N*N*N)*sizeof(float), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_b not copied.\n");
    // set up n on device
    cudaStatus = hipMalloc((void**)&d_n, sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("d_n not allocated.\n");
    int n = N;
    cudaStatus = hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("d_n not copied.\n");
    // finally, run on gpu
    hipEventRecord(start);
    tiled <<< N*N, N >>> (d_a, d_b, N);
    hipEventRecord(stop);
    // copy result back to host
    float * a = new float[N*N*N+1];
    cudaStatus = hipMemcpy(a, d_a, (N*N*N)*sizeof(float), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess)
        printf("a not returned to host.\n");
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(PRINT_RUNTIME==1) printf("Nontiled took %f milliseconds.\n",milliseconds);
    hipFree(d_a); hipFree(d_b);
    return a;
}

int main(int argc, char* argv[]){
    float * b;
    b = (float *)malloc(N*N*N*sizeof(float));
    for(int i=0; i<N; i++) 
        for(int j=0; j<N; j++)
            for(int k=0; k<N; k++)
                b[i+j+k]= static_cast <float> (rand())/(static_cast <float> (RAND_MAX/RANGE));
    float * a;
    a = q2_nontiled(b);
    if(PRINT_RESULT==1)
        for(int i=0; i<N; i++)
            for(int j=0; j<N; j++)
                for(int k=0; k<N; k++)
                    if(a[i+j+k]>0.) printf("[%d,%d,%d] : %f\n", i+1, j+1, k+1, a[i+j+k]); 
    free(b);
    free(a);
    b = (float *)malloc(N*N*N*sizeof(float));
    for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            for(int k=0; k<N; k++)
                b[i+j+k]= static_cast <float> (rand())/(static_cast <float> (RAND_MAX/RANGE));
    a = q2_tiled_shared_memory(b);
    if(PRINT_RESULT==1)
        for(int i=0; i<N; i++)
            for(int j=0; j<N; j++)
                for(int k=0; k<N; k++)
                    if(a[i+j+k]>0.) printf("[%d,%d,%d] : %f\n", i+1, j+1, k+1, a[i+j+k]);
    free(b);
    free(a); 
    b = (float *)malloc(N*N*N*sizeof(float));
    for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            for(int k=0; k<N; k++)
                b[i+j+k]= static_cast <float> (rand())/(static_cast <float> (RAND_MAX/RANGE));
    a = q2_tiled(b);
    if(PRINT_RESULT==1)
        for(int i=0; i<N; i++)
            for(int j=0; j<N; j++)
                for(int k=0; k<N; k++)
                    if(a[i+j+k]>0.) printf("[%d,%d,%d] : %f\n", i+1, j+1, k+1, a[i+j+k]);
    free(b);
    free(a);
    return 0;
}
