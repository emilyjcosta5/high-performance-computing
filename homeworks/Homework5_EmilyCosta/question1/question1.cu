#include <cmath>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BIN_COUNT 128
#define BLOCK_SIZE 512
#define N 33554432
#define RANGE 1000000
#define PRINT_RESULT 0
#define PRINT_RUNTIME 1

__global__ void kernel(const int *numbers, int *histogram, const int *bin_count){
     /*__shared__ unsigned int local[NUM_THREADS];
     local[threadIdx.x] = 0;
     __syncthreads();
     int offset = blockDim.x * gridDim.x; // to move to next block
     //int bin_size = (int)(RANGE/NUM_THREADS);
     int bin = 0;
     for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<size; i+=offset){
         bin = (int)(numbers[i]/bin_size);     
         atomicAdd(&local[bin], 1);
     }
     __syncthreads();
    // since we inferred each thread has one bin
    atomicAdd(&(histogram[threadIdx.x]), local[threadIdx.x]);*/
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int num = numbers[i];
    int bin = num % *bin_count;
    atomicAdd(&(histogram[bin]), 1);
}

int* histogram_gpu(int* numbers){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int num_blocks = N/BLOCK_SIZE;
    if(N%BLOCK_SIZE!=0) num_blocks++;
    // allocate memory for histogram
    int* dev_histogram;
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&dev_histogram, (BIN_COUNT+1)*sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("dev_histogram not allocated.\n");
    // allocate memory for numbers
    int* dev_numbers;
    cudaStatus = hipMalloc((void**)&dev_numbers, (N+1)*sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("dev_numbers not allocated.\n");
    // set histogram values into the device
    cudaStatus = hipMemset(dev_histogram, 0, (BIN_COUNT+1) * sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("dev_histogram not memset.\n");
    // copy numbers into the device
    cudaStatus = hipMemcpy(dev_numbers, numbers, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("dev_numbers not copied.\n");
    int * host_numbers = new int[N+1];
    hipMemcpy(host_numbers, dev_numbers, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
    if(host_numbers[0]!=numbers[0]) printf("Numbers not copied correctly.\n");
    int* dev_bin_count;
    cudaStatus = hipMalloc((void**)&dev_bin_count, sizeof(int));
    if(cudaStatus != hipSuccess)
        printf("dev_bin_count not allocated.\n");
    int bin_count = BIN_COUNT;
    cudaStatus = hipMemcpy(dev_bin_count, &bin_count, sizeof(int), hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess)
        printf("dev_bin_count not copied.\n");
    // finally, run on gpu
    hipEventRecord(start);
    kernel <<< num_blocks , BLOCK_SIZE >>> (dev_numbers, dev_histogram, dev_bin_count);
    hipEventRecord(stop);
    // copy result back to host
    int* histogram = new int[BLOCK_SIZE+1];
    cudaStatus = hipMemcpy(histogram, dev_histogram, (BIN_COUNT+1)*sizeof(int), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess)
        printf("histogram not returned to host.\n");
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if(PRINT_RUNTIME==1) printf("Total gpu runtime (milliseconds): %f\n", milliseconds);
    hipFree(dev_histogram); hipFree(dev_numbers);
    return histogram;
}

int main(int argc, char* argv[]){
    int numbers[N];
    for(int i=0; i<N; i++) numbers[i]=(rand()%RANGE)+1;
    int* histogram;
    histogram = histogram_gpu(numbers);
    if(PRINT_RESULT==1)
        for (int i=0; i<BIN_COUNT; i++)
            printf("%d : %d\n", i+1, histogram[i]); 
    delete [] histogram;
    return 0;
}
