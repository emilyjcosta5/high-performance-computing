#include<cmath>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_BINS 10
#define GRID_SIZE 32 /* should be a multiple of 32 */
#define N 1000

__global__ void kernel_histogram(unsigned char* numbers, long size, unsigned int* histogram, int num_bins){
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    if(tid>=size) return;
    unsigned char number = numbers[tid];
    int bin = number % num_bins;
    atomicAdd(&histogram[bin], 1);
}


